
#include <hip/hip_runtime.h>
// __device__ bool isPointInTrapezoid(float px, float py, float *trapezoid) {
//     // Implement the point-in-trapezoid logic here, similar to the point-in-polygon test in the provided C++ code
//     // The 'trapezoid' parameter is an array of coordinates defining the trapezoid
//     // This function returns true if the point (px, py) is inside the trapezoid
// }

// __global__ void findPointsBehindWallsKernel(float3 *points, int numPoints, float *wallPoses, int numWalls, bool *output) {
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if (index < numPoints) {
//         float3 point = points[index];
//         bool isBehind = false;

//         // Each wall is represented by two points in 'wallPoses'. Thus 'numWalls' should be half the length of 'wallPoses'
//         for (int i = 0; i < numWalls; i += 2) {
//             float trapezoid[8]; // Store the trapezoid coordinates here
//             // Calculate trapezoid based on wallPoses[i] and wallPoses[i+1]
//             // You need to implement the trapezoid extension similar to your C++ 'createPolygon' function
            
//             if (isPointInTrapezoid(point.x, point.y, trapezoid)) {
//                 isBehind = true;
//                 break;
//             }
//         }
//         output[index] = isBehind;
//     }
// }

// // Host code to allocate memory, copy data, and launch the kernel
// // You need to set up `points`, `wallPoses`, and allocate memory for `output`
// // Then, you copy these data to device memory, call the kernel, and copy `output` back to host to check which points to remove
